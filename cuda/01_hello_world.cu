
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void fun() {
    printf("hello world from GPU\n");
}

int main() {
    printf("hello world from CPU\n");

    // fun<<< m,n >>>()：代表一个kernel被调用的固定格式。m,n分别代表块和线程数。后续文章会进行介绍。
    fun<<<2, 3>>>();
    hipDeviceSynchronize();
}