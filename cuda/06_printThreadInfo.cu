#include "hip/hip_runtime.h"
// https://zhuanlan.zhihu.com/p/455652400
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "learnCuda.h"

void sumArrayonHost(float* A, float* B, float* C, const int N) {
    for(int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

// __global__ void printThreadIndex(int* A, const int nx, const int ny) {
//     //1、先定位线程坐标
//     int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     int iy = threadIdx.y + blockIdx.y * blockDim.y;

//     //2、然后定位对应数据的内存坐标（因为内存是线性排列，所以是  x + y*nx
//     // thread Id, block Id, 矩阵元素的坐标, global memory的偏移量, 即内存线性空间的坐标, 元素值
//     unsigned int idx = iy * nx + ix;
//     printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d) "
//            "global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
// }

int main(int arc, char** argv) {
    printf("%s Starting...\n", argv[0]);
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;

    size_t nBytes = nxy * sizeof(float);

    // malloc host memory
    int* h_A = (int*)malloc(nBytes);
    initialInt(h_A, nxy);
    printMatrix(h_A, nx, ny);

    // malloc device memory
    int* d_A;
    hipMalloc((void**)&d_A, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    
    // set up execution configration
    dim3 block(4,2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    printThreadIndex<<<grid, block>>>(d_A, nx, ny);
    hipDeviceSynchronize();

    // free host and device memory
    hipFree(d_A);
    free(h_A);
    hipDeviceReset();
    return 0;
}