#include "hip/hip_runtime.h"
#include <stdio.h>

const int N = 32*1024*1024;           // 输入矩阵的大小input size 32M的global memory
const int thread_per_block = 256;    // 每个block的thread数  2^8个
const int block_num = N / thread_per_block;                

__global__ void reduce3(float* d_in, float* d_out) {
    __shared__ float cache[block_num];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2);

    cache[tid] = d_in[] + ; 


}

// warp是并行计算时最小的并发结构，通常由连续32个thread组成。
// 同一个warp中的所有thread是完全并行的，且必须要执行相同的指令，当同一warp中的thread分配了不同的指令时，会发生warp divergence
// 若一个warp（32个线程）里不同线程因为分支条件执行了不同指令，效率会极低。同时 % 取余运算的效率也很低

// 没写错，这里临时变量就是为了找起始下标，线程是连着编号的，但是每个线程操作的元素第一个下标需要计算
// 图没错，每个S的取值代表纵轴一个step。 每个tid的取值代表横轴的一个thread
__global__ void reduce1(float* d_in, float* d_out) {

    __shared__ float cache[thread_per_block];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    cache[tid] = d_in[i];
    __syncthreads();
    
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if(index < blockDim.x) {
            cache[index] += cache[index + s];
        }
        __syncthreads();
    }

    if(tid ==0) {
        d_out[blockIdx.x] = cache[tid];
    }
}

__global__ void reduce1_1(float* d_in, float* d_out) {
    __shared__ float cache[thread_per_block];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    cache[tid] = d_in[i];
    __syncthreads();
    
    for(unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if(tid < stride) {
            cache[tid] += cache[tid + stride];
        }
        __syncthreads();
    }

    if(tid ==0) {
        d_out[blockIdx.x] = cache[tid];
    }
}

__global__ void reduce0(float* d_in, float* d_out) {
    // 分配shared memory ，将数据global load到shared memory
    __shared__ float cache[thread_per_block];

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int cacheIndex = threadIdx.x;

    // each thread load one element form global to shared mem
    cache[cacheIndex] = d_in[tid];
    __syncthreads();

    // do reduction in each block
    for(unsigned int i = 1; i < blockDim.x; i *= 2) { // or i < thread_per_block
        if(cacheIndex % (2 * i) == 0) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }     
        __syncthreads();
    }

    if(cacheIndex == 0) {
        d_out[blockIdx.x] = cache[0]; 
    }
}

bool check(float* res_h, float* ans) {
    for(int i = 0; i < block_num; i++) {
        if(res_h[i] != ans[i]) {
            printf("Wrong Answer\n");
            return false;
        }
    }
    printf("Right Answer\n");
    return true;
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out;
    // cpu上分配内存
    h_in = (float*)malloc(N * sizeof(float));
    h_out = (float*)malloc(block_num * sizeof(float)); // 这里是只算第一级block输出的，不算第二级的了
    for(int i = 0; i < N; i++) {
        h_in[i] = 1;
    }

    // gpu上分配内存
    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, block_num * sizeof(float));

    // 拷贝数据从cpu到gpu
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, N * sizeof(float), hipMemcpyHostToDevice);


    dim3 Grid(block_num, 1);           // 一个grid里多少个block
    dim3 Block(thread_per_block, 1);   // 一个block里多少个thread
    // reduce0<<<Grid, Block>>>(d_in, d_out);
    reduce1_1<<<Grid, Block>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

    float* ans = (float*)malloc(block_num * sizeof(float));
    for(int i = 0; i < block_num; i++) {
        float curr = 0;
        for(int j = 0; j < thread_per_block; j++) {
            curr += h_in[i * thread_per_block + j];
        }
        ans[i] = curr;
    }

    if(check(h_out, ans) == false) {
        for(int i = 0; i < 1024; i++) {
            if(h_out[i] != ans[i]) {
                printf("%d, %lf\n", i, h_out[i]);
            }
            // printf("%lf", h_out[i]);
        }
        printf("\n");
    }
    printf("block_num: %d\n", block_num);

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;


}
