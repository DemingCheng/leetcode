#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_start.h"

// CPU 对照组，对比加速比
void sumMatrix2DonCPU(float* MatA, float* MatB, float* MatC, int nx, int ny) {
    float* a = MatA;
    float* b = MatB;
    float* c = MatC;


}   

__global__ void sumMatrix(float* MatA, float* MatB, float* MatC, int nx, int ny) {

}

int main() {
    //设备初始化
    printf("strating...\n");
    initDevice(0);

    // 二维矩阵大小 4096*4096 单精度浮点型
    int nx = 1<<12; // 4096 2^12 位运算-移位
    int ny = 1<<12; // 4096 
    int nBytes = nx * ny * sizeof(float);

    // Malloc 开辟主机内存
    float* A_host = (float*)malloc(nBytes);
    float* B_host = (float*)malloc(nBytes);
    float* C_host = (float*)malloc(nBytes);  
    float* C_from_gpu = (float*)malloc(nBytes);
    initialData(A_host, nx*ny);
    initialData(B_host, nx*ny);

    // hipMalloc 开辟device内存
    float* A_dev = NULL;
    float* B_dev = NULL;
    float* C_dev = NULL;  
    CHECK(hipMalloc((void**)&A_dev), nBytes);
    CHECK(hipMalloc((void**)&B_dev), nBytes);
    CHECK(hipMalloc((void**)&C_dev), nBytes);

    // 输入数据从host内存拷贝到device内存
    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));

    // 二维线程块 32*32
    dim3 block(32, 32);
    // 二维线程网络 128*128
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);



}