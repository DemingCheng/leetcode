
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 32*1024*1024;           // 输入矩阵的大小input size 32M的global memory
const int thread_per_block = 256;    // 每个block的thread数  2^8个
const int block_num = N / thread_per_block;                  

// __global__ void reduce3(float* d_in, float* d_out) {
//     __shared__ float cache[block_num];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2);

//     cache[tid] = d_in[] + ; 


// }

// warp是并行计算时最小的并发结构，通常由连续32个thread组成。
// 同一个warp中的所有thread是完全并行的，且必须要执行相同的指令，当同一warp中的thread分配了不同的指令时，会发生warp divergence
// 若一个warp（32个线程）里不同线程因为分支条件执行了不同指令，效率会极低。同时 % 取余运算的效率也很低

// 没写错，这里临时变量就是为了找起始下标，线程是连着编号的，但是每个线程操作的元素第一个下标需要计算
// 图没错，每个S的取值代表纵轴一个step。 每个tid的取值代表横轴的一个thread
__global__ void reduce1(float* d_in, float* d_out) {

    __shared__ float cache[thread_per_block];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    cache[tid] = d_in[i];
    __syncthreads();
    
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if(index < blockDim.x) {
            cache[index] += cache[index + s];
        }
        __syncthreads();
    }

    if(tid ==0) {
        d_out[blockIdx.x] = cache[tid];
    }
}

__global__ void reduce1_1(float* d_in, float* d_out) {
    __shared__ float cache[thread_per_block];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    cache[tid] = d_in[i];
    __syncthreads();
    
    for(unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if(tid < stride) {
            cache[tid] += cache[tid + stride];
        }
        __syncthreads();
    }

    if(tid ==0) {
        d_out[blockIdx.x] = cache[tid];
    }
}

// 最基础reduction
__global__ void reduce_0(float* d_in, float* d_out) {
    // 分配shared memory ，将数据global load到shared memory
    __shared__ float sdata[thread_per_block];

    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tid = threadIdx.x;  // 块内index

    // each thread load one element form global to shared mem
    sdata[tid] = d_in[i];
    __syncthreads();

    // do reduction in each block
    for(unsigned int stride = 1; stride < blockDim.x; stride *= 2) { // or stride < thread_per_block
        if(tid % (2 * stride) == 0) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) {
        d_out[blockIdx.x] = sdata[tid]; 
    }
}

__global__ void reduce0(float* d_in,float* d_out){
    __shared__ float sdata[thread_per_block];

    //each thread loads one element from global memory to shared mem
    unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
    unsigned int tid=threadIdx.x;
    sdata[tid]=d_in[i];
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=1; s<blockDim.x; s*=2){
        if(tid%(2*s) == 0){
            sdata[tid]+=sdata[tid+s];
        }
        __syncthreads();
    }
    
    // write result for this block to global mem
    if(tid==0) {
        d_out[blockIdx.x]=sdata[tid];
        printf("blockIdx.x: %lf", d_out[blockIdx.x]);
    }
}

bool check(float* res_h, float* ans) {
    for(int i = 0; i < block_num; i++) {
        if(res_h[i] != ans[i]) {
            printf("Wrong Answer\n");
            return false;
        }
    }
    printf("Right Answer\n");
    return true;
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out;
    // cpu上分配内存
    h_in = (float*)malloc(N * sizeof(float));
    h_out = (float*)malloc(block_num * sizeof(float)); // 这里是只算第一级block输出的，不算第二级的了
    for(int i = 0; i < N; i++) {
        h_in[i] = 1;
    }

    // gpu上分配内存
    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, block_num * sizeof(float));

    // 拷贝数据从cpu到gpu
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, N * sizeof(float), hipMemcpyHostToDevice);


    dim3 Grid(block_num, 1);           // 一个grid里多少个block
    dim3 Block(thread_per_block, 1);   // 一个block里多少个thread
    // reduce0<<<Grid, Block>>>(d_in, d_out);
    reduce0<<<Grid, Block>>>(d_in, d_out);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Failed: Cuda error %s:%d '%s'\n", __FILE__,__LINE__,hipGetErrorString(err));
    }
    

    hipMemcpy(h_out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

    float* ans = (float*)malloc(block_num * sizeof(float));
    for(int i = 0; i < block_num; i++) {
        float curr = 0;
        for(int j = 0; j < thread_per_block; j++) {
            curr += h_in[i * thread_per_block + j];
        }
        ans[i] = curr;
    }

    // if(check(h_out, ans) == false) {
    //     for(int i = 0; i < 1024; i++) {
    //         if(h_out[i] != ans[i]) {
    //             printf("%d, %lf\n", i, h_out[i]);
    //         }
    //         // printf("%lf", h_out[i]);
    //     }
    //     printf("\n");
    // }
    
    printf("check(h_out, ans): %s\n", check(h_out, ans) ? "true": "false");
    printf("block_num: %d\n", block_num);

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;


}
