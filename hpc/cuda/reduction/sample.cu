
#include <hip/hip_runtime.h>
#include <stdio.h>

#define min(a, b) (a<b?a:b)

const int N = 3000;
const int threadsPerBlock = 1024;
const int blocksPerGrid = 1;// min(32, (N+threadsPerBlock-1)/threadsPerBlock);

// 若涉及线程频读写，将数据常驻shared memory，提高运行效率。同一block的thread可以共享该shared memory
// cuda中使用__shared__关键字，这里使用__syncthreads()控制线程同步。多个thread对shared memory访问，需要对线程进行同步操作，避免竞争

__global__ void dot(float* a, float* b, float* c){
    // 分配shared memory
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

    // 
	float temp = 0;
	while(tid < N){
		temp += a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	//对线程块中的线程进行同步
	__syncthreads();

	int i = blockDim.x/2;

	while(i != 0){
		if(cacheIndex < i){
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0){
		c[blockIdx.x] = cache[0];
		//printf("%f\n", c[blockIdx.x]);
	}
}

int main(){

	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	//在CPU上面分配内存
	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	//在GPU上分配内存
	hipMalloc((void**)&dev_a, N*sizeof(float));
	hipMalloc((void**)&dev_b, N*sizeof(float));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float));

	//填充主机内存
	for(int i = 0; i < N; i++){
		a[i] = 1;
		b[i] = 0;
	}

	//将 数组a 和 数组b 复制到GPU
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

	//将数组 dev_partial_c 从 GPU 复制到 CPU
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

	//在CPU上完成最终的求和运算
	c = 0.0;
	for(int i = 0; i < blocksPerGrid; i++){
		c += partial_c[i];
	}

	printf("%s\n", "======================================");
	// for(int i = 0; i < N; i++){
	// 	printf("%f  %f \n", a[i], b[i]);
	// }

	printf("c = %f \n", c);
	printf("blocksPerGrid %d \n", blocksPerGrid);

	// 释放 GPU 上的内存
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	// 释放 CPU 上的内存
	free(a);
	free(b);
	free(partial_c);

	return 0;
}
