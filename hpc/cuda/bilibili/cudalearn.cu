#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_WIDTH 16

__global__ void MatrixMul(float* M_d, float* N_d, float* P_d, const int WIDTH) {   // Q:为什么这里是一维指针，而cudaMalloc里是二维指针 cudaError_t cudaMalloc(void** devPtr, size_t size); 在设备上分配size字节的线性内存，并返回分配内存的指针*devPtr。
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    for(int k = 0; k < WIDTH; k++) {
        P_d[row * WIDTH + col] += M_d[row * WIDTH + k] * N_d[k * WIDTH + col]; 
    }
}

int main() {
    const int WIDTH = 512;
    // 定义CPU矩阵
    float M_h[WIDTH][WIDTH], N_h[WIDTH][WIDTH], P_h[WIDTH][WIDTH];
    // 定义GPU矩阵
    float* M_d, *N_d, *P_d;
    // 初始化host上的矩阵
    for(int i = 0; i < WIDTH; i++) {
        for(int j = 0; j < WIDTH; j++) {
            M_h[i][j] = 1;
            N_h[i][j] = 2;
        }
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elaspedTime;

    // 在GPU上分配空间
    int size = WIDTH*WIDTH*sizeof(int);
    hipMalloc((void**)&M_d, size);
    hipMalloc((void**)&N_d, size);
    hipMalloc((void**)&P_d, size); 

    // 将CPU上的数据传输到GPU的显存中 Q:显存？
    hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);

    // 设置kernel的执行设置
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH, 1);

    // 执行kernel函数
    MatrixMul<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, WIDTH);

    // 释放device上空间
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    return 0;
}