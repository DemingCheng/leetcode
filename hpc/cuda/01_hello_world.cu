
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void fun() {
    int tid = threadIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    printf("hello world from GPU, tid: %d, i: %d\n", tid, i);
}

__global__ void checkIndex(void)
{
  printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)\
  gridDim(%d,%d,%d)\n",threadIdx.x,threadIdx.y,threadIdx.z,
  blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,
  gridDim.x,gridDim.y,gridDim.z);
}

int main() {
    printf("hello world from CPU\n");

    // fun<<< m,n >>>()：代表一个kernel被调用的固定格式。m,n分别代表块和线程数。后续文章会进行介绍。
    // fun<<<2, 3>>>();

    int nElem=6;
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
    checkIndex<<<grid,block>>>();   
    
    hipDeviceSynchronize();
}

