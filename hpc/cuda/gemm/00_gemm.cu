#include "hip/hip_runtime.h"
// https://zhuanlan.zhihu.com/p/98654665
#include <stdlib.h>
#include <stdio.h>

#define N 1024
#define thread_per_block 256
#define block_num 32

// const int N =1024;


__global__ void gemm_0(float* d_A, float* d_B, float* d_C, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    float sum = 0.0;

    for(int k = 0; k < n; k++) {
        sum += d_A[row * n + k] * d_B[k * n + col];
    }

    d_C[row * n + col] = sum;
    printf(d_C[row * n + col]);
}

// __global__ void gemm_0(float* d_A, float* d_B, float* d_C, int N) {
// }

void prinfMatrix(float* mat, int row, int col) {
    for(int i = 0; i < row; i++) {
        for(int j = 0; j < col; j++) {
            printf("%lf ", mat[i * row + j]);
        }
        printf("/n");
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(N * N * sizeof(float));
    h_B = (float*)malloc(N * N * sizeof(float));
    h_C = (float*)malloc(N * N * sizeof(float));

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            h_A[i * N + j] = 1;
            h_B[i * N + j] = 2;
        }
    }

    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(thread_per_block, 1);
    dim3 grid(block_num, 1);

    gemm_0<<<grid, block>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // prinfMatrix(h_C, N, N);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}